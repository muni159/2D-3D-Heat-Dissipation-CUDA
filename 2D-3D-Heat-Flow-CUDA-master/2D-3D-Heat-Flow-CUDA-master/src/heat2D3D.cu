#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<string>
#include<algorithm>
#include<vector>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<iomanip>
#include <limits>
#define BLOCK_SIZE 8
using namespace std;
__global__ void kernel(float *Told,float *Tnew,int *dim,float *k_val,float *sp,int *grid_w,int *grid_h,int *grid_d,int *pn) 
{
 if (*dim==2)
	{
		__shared__ float Mat[BLOCK_SIZE+2][BLOCK_SIZE+2];
		int j = blockIdx.x * blockDim.x + threadIdx.x;
		int i = blockIdx.y * blockDim.y + threadIdx.y;
		int jmat = threadIdx.x+1;
		int imat = threadIdx.y+1;

		if (i<=*grid_h && j<=*grid_w)
		{
			int lengthj=(blockIdx.x==(int)*grid_w/BLOCK_SIZE)? *grid_w%BLOCK_SIZE :BLOCK_SIZE;
			int lengthi=(blockIdx.y==(int)*grid_h/BLOCK_SIZE)? *grid_h%BLOCK_SIZE :BLOCK_SIZE;
			Mat[imat][jmat]=Told[i*(*grid_h)+j];

			if (threadIdx.x < 1)
			{
				Mat[imat][jmat-1]=(j<1)?  *(Told+i*(*grid_h)+(j)): *(Told+i*(*grid_h)+j-1);  /// check j or j-1
				Mat[imat][jmat + lengthj]=(j >= *grid_w-lengthj)? *(Told+i*(*grid_h)+j+lengthj-1):*(Told+i*(*grid_h)+j+lengthj);	
			}

			 if (threadIdx.y < 1)
			{
				Mat[imat-1][jmat]= (i < 1)? *(Told+(i)*(*grid_h)+j): *(Told+(i-1)*(*grid_h)+j);  /// check j or j-1
				Mat[imat + lengthi][jmat]=(i >= (*grid_h)-lengthi)? *(Told+(i+lengthi-1)*(*grid_h)+j): *(Told+(i+lengthi)*(*grid_h)+j);
			}
			__syncthreads();
			
			if (i<*grid_h && j<*grid_w)
			{
				*(Tnew+i*(*grid_h)+j)=Mat[imat][jmat]+(*k_val)*(Mat[imat+1][jmat]+Mat[imat-1][jmat]+Mat[imat][jmat+1]+Mat[imat][jmat-1]-4*Mat[imat][jmat]);
			
				for (int ii = 0; ii < *pn; ii+=5)
				{
					if (j>=(int)sp[ii]&&i>=(int)sp[ii+1]&&j<((int)sp[ii]+(int)sp[ii+2])&&i<((int)sp[ii+1]+(int)sp[ii+3]))
					{
						*(Tnew+i*(*grid_h)+j)=(float)sp[ii+4];
					}
				}
			*(Told+i*(*grid_h)+j)=*(Tnew+i*(*grid_h)+j);
			}
		}

	}
	else if (*dim ==3)
	{   
		__shared__ float Mat[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
		int j = blockIdx.x * blockDim.x + threadIdx.x;
		int i = blockIdx.y * blockDim.y + threadIdx.y;
		int k = blockIdx.z * blockDim.z + threadIdx.z;
		int jmat = threadIdx.x+1;
		int imat = threadIdx.y+1;
		int kmat = threadIdx.z+1;

		if (i<=*grid_h && j<=*grid_w && k<=*grid_d)
		{
			int lengthj=(blockIdx.x==(int)*grid_w/BLOCK_SIZE)? *grid_w%BLOCK_SIZE :BLOCK_SIZE;
			int lengthi=(blockIdx.y==(int)*grid_h/BLOCK_SIZE)? *grid_h%BLOCK_SIZE :BLOCK_SIZE;
			int lengthk=(blockIdx.z==(int)*grid_d/BLOCK_SIZE)? *grid_d%BLOCK_SIZE :BLOCK_SIZE;
			Mat[imat][jmat][kmat]=Told[i*(*grid_h)*(*grid_d)+j*(*grid_d)+k];

			if (threadIdx.x < 1)
			{
				Mat[imat][jmat-1][kmat]=(j<1)?  *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k): *(Told+i*(*grid_h)*(*grid_d)+(j-1)*(*grid_d)+k);  /// check j or j-1
				Mat[imat][jmat + lengthj][kmat]=(j >= *grid_w-lengthj)? *(Told+i*(*grid_h)*(*grid_d)+(j+lengthj-1)*(*grid_d)+k):*(Told+i*(*grid_h)*(*grid_d)+(j+lengthj)*(*grid_d)+k);
			}

			 if (threadIdx.y < 1)
			{
				Mat[imat-1][jmat][kmat]= (i < 1)? *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k): *(Told+(i-1)*(*grid_h)*(*grid_d)+j*(*grid_d)+k);  /// check j or j-1
				Mat[imat + lengthi][jmat][kmat]=(i >= *grid_h-lengthi)? *(Told+(i+lengthi-1)*(*grid_h)*(*grid_d)+j*(*grid_d)+k): *(Told+(i+lengthi)*(*grid_h)*(*grid_d)+j*(*grid_d)+k);	
				
			}
			 if (threadIdx.z < 1)
			{
				Mat[imat][jmat][kmat-1]= (k < 1)? *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k): *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k-1);  /// check j or j-1
				Mat[imat][jmat][kmat+lengthk]= (k >= *grid_d-lengthk)? *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k+lengthk-1): *(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k+lengthk);	
			}
			__syncthreads();
			
			if (i<*grid_h && j<*grid_w && k<*grid_d)
			{
				*(Tnew+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k)=Mat[imat][jmat][kmat]+(*k_val)*(Mat[imat+1][jmat][kmat]+Mat[imat-1][jmat][kmat]+Mat[imat][jmat+1][kmat]+Mat[imat][jmat-1][kmat]+Mat[imat][jmat][kmat-1]+Mat[imat][jmat][kmat+1]-6*Mat[imat][jmat][kmat]);
			
				for (int ii = 0; ii < *pn; ii+=7)
				{
					if (j>=(int)sp[ii]&&i>=(int)sp[ii+1]&&k>=(int)sp[ii+2]&&j<((int)sp[ii]+(int)sp[ii+3])&&i<((int)sp[ii+1]+(int)sp[ii+4])&&k<((int)sp[ii+2]+(int)sp[ii+5]))
					{
						*(Tnew+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k)=(float)sp[ii+6];
					}
				}
			*(Told+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k)=*(Tnew+i*(*grid_h)*(*grid_d)+j*(*grid_d)+k);
			}
		}
	}
}

int main(int argc, char const *argv[])
{

	if (argc!=2)
	{
		cout<<"There seems to be a problem with number of arguments!"<<endl;
		exit(1);
	}
	// parameters
	vector <string> pstring;
	// Parsing the conf file values
	ifstream input_file(argv[1]);
	string Line;
	int paramter_number=0;
	string temp;
	int size=0;
	while(getline(input_file,Line))
	{
		if (Line.length()==0)
		{
			continue;
		}
			for (int i = 0; i < Line.length() ; i++)
			{
				if (Line[i]==' '||(int)Line[i]==13||(int)Line[i]==9)
				{
					continue;	
				}
				else
				{
					if (Line.at(i)=='#')
					{
						break;
					}
					if (Line.at(i)==',')
					{
						pstring.push_back(temp);
						temp.clear();

						size=0;
						paramter_number+=1;
						continue;
					}
					temp.push_back(Line.at(i));	
					size+=1;						
				}
			}
			if (temp.length()!=0)
			{
			pstring.push_back(temp);
			temp.clear();
			size=0;
			paramter_number+=1;
			}
	}
	input_file.close();

	// Getting all values
	int dimension=(int)(pstring[0].at(0))-48;
	pstring[1].insert(0,1,'0');
	float k=atof(pstring[1].c_str());
	int timestep = atoi(pstring[2].c_str());
	std::vector<float> source_parameters;

	int grid_width,grid_height,grid_depth;
	float start_temp;
	/*cout<<"\nNumber of Dimensions : "<<dimension;
	cout<<"\nK                    : "<<k;
	cout<<"\nTimestep             : "<<timestep;*/
	float *d_Told,*d_Tnew,*d_k_val,*d_sp;
	int *d_dim,*d_grid_w,*d_grid_h,*d_grid_d,*pn;

	if (dimension==2)
	{
		grid_width= atoi(pstring[3].c_str());
		grid_height=atoi(pstring[4].c_str());
		start_temp=atof(pstring[5].c_str());
		/*cout<<"\nGrid Width           : "<<grid_width;
		cout<<"\nGrid Height          : "<<grid_height;
		cout<<"\nStarting Temperature : "<<start_temp<<endl;*/
		for (int i = 6; i < paramter_number; i++)
		{
			source_parameters.push_back(atof(pstring[i].c_str()));
		}
		/*for (int i = 0; i < paramter_number-6; i=i+5)
		{
			cout<<"\nX : "<<source_parameters[i]<<" Y : "<<source_parameters[i+1];
			cout<<" Width : "<<source_parameters[i+2]<<" Height : "<<source_parameters[i+3];
			cout<<" Temperature : "<<source_parameters[i+4]<<endl;
		}*/
		int length = (grid_width>=grid_height)? grid_width*grid_width:grid_height*grid_height;
		float Told[length] = {0};
		float Tnew[length] = {0}; 
		int size_T = (length)*sizeof(float);

		for (int i = 0; i < grid_height; i++)
		{
			for (int j = 0; j < grid_width; j++)
			{
				for (int ii = 0; ii < (paramter_number-6); ii+=5)
				{
					if (j>=(int)source_parameters[ii]&&i>=(int)source_parameters[ii+1]&&j<((int)source_parameters[ii]+(int)source_parameters[ii+2])&&i<((int)source_parameters[ii+1]+(int)source_parameters[ii+3]))
					{
						Told[i*grid_height+j]=source_parameters[ii+4];
						break;
					}
					else
					{
						Told[i*grid_height+j]=start_temp;
					}
				}
			}
		}
		int p_n=paramter_number-6;
		float* sp = &source_parameters[0];

		hipMalloc((void **)&d_Told, size_T);
		hipMalloc((void **)&d_Tnew, size_T);
		hipMalloc((void **)&d_k_val,sizeof(float));
		hipMalloc((void **)&d_dim,sizeof(int));
		hipMalloc((void **)&d_sp,p_n*sizeof(float));
		hipMalloc((void **)&d_grid_w,sizeof(int));
		hipMalloc((void **)&d_grid_h,sizeof(int));
		hipMalloc((void **)&d_grid_d,sizeof(int));
		hipMalloc((void **)&pn,sizeof(int));

		hipMemcpy(d_Told,Told,size_T,hipMemcpyHostToDevice);
		hipMemcpy(d_Tnew,Tnew,size_T,hipMemcpyHostToDevice);
		hipMemcpy(d_k_val,&k,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d_dim,&dimension,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_sp,sp,p_n*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_w,&grid_width,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_h,&grid_height,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_d,&grid_depth,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(pn,&p_n,sizeof(int),hipMemcpyHostToDevice);

		dim3 Block(BLOCK_SIZE,BLOCK_SIZE);
		dim3 Grid((int)ceil((grid_width+BLOCK_SIZE-1)/BLOCK_SIZE),(int)ceil((grid_height+BLOCK_SIZE-1)/BLOCK_SIZE));

		for (int t = 0; t <timestep; t++)
		{	
			kernel<<<Grid,Block>>>(d_Told,d_Tnew,d_dim,d_k_val,d_sp,d_grid_w,d_grid_h,d_grid_d,pn);
		}
		hipMemcpy(Tnew,d_Tnew,size_T,hipMemcpyDeviceToHost);
		hipFree(d_Tnew);hipFree(d_Told);hipFree(d_k_val);hipFree(d_dim);hipFree(d_sp);hipFree(d_sp);
		hipFree(d_grid_d);hipFree(d_grid_h);hipFree(d_grid_w);
		/*cout<<"\nTnew: \n";
		for (int i = 0; i < grid_height; i++)
		{
			for (int j = 0; j < grid_width-1; j++)
			{
				cout<<left<<Tnew[i*grid_height+ j]<<", ";
				//printf("%f, ",Tnew[i][j] );
			}
			cout<<left<<Tnew[i*grid_height + grid_width-1]<<endl;
			//printf("%f,\n",Tnew[i][grid_width-1] );
		}*/
		ofstream build ("heatOutput.csv", std::ofstream::out);
		for (int i = 0; i < grid_height; i++)
		{
			for (int j = 0; j < grid_width-1; j++)
			{
				build<<std::setprecision(6)<<Tnew[i*grid_height+ j]<<", ";
			}
			build<<std::setprecision(6)<<Tnew[i*grid_height+ grid_width-1];
			if (i!=grid_height-1)
			{
				build<<"\n";
			}
		}
		build.close();
	}
	else if (dimension==3)
	{
		grid_width= atoi(pstring[3].c_str());
		grid_height=atoi(pstring[4].c_str());
		grid_depth=atoi(pstring[5].c_str());
		start_temp=atof(pstring[6].c_str());
		/*cout<<"\nGrid Width           : "<<grid_width;
		cout<<"\nGrid Height          : "<<grid_height;
		cout<<"\nGrid Depth           : "<<grid_depth;
		cout<<"\nStarting Temperature : "<<start_temp<<endl;*/
		for (int i = 7; i < paramter_number; i++)
		{
			source_parameters.push_back(atof(pstring[i].c_str()));
		}
		/*for (int i = 0; i < paramter_number-7; i=i+7)
		{
			cout<<"\nX : "<<source_parameters[i]<<" Y : "<<source_parameters[i+1]<<" Z : "<<source_parameters[i+2] ;
			cout<<" Width : "<<source_parameters[i+3]<<" Height : "<<source_parameters[i+4]<< " Depth : "<<source_parameters[i+5];
			cout<<" Temperature: "<<source_parameters[i+6]<<endl;
		}*/
		int length = (grid_width>=grid_height)?((grid_width>=grid_depth)?grid_width*grid_width*grid_width:grid_depth*grid_depth*grid_depth):((grid_height>=grid_depth)?grid_height*grid_height*grid_height:grid_depth*grid_depth*grid_depth);
		float Told[length] = {0};
		float Tnew[length] = {0}; 
		int size_T = (length)*sizeof(float);

		for (int kk = 0; kk < grid_depth; kk++)
			{
			for (int i = 0; i < grid_height; i++)
				{
					for (int j = 0; j < grid_width; j++)
					{
						for (int ii = 0; ii < (paramter_number-7); ii+=7)
						{
							if (j>=(int)source_parameters[ii]&&i>=(int)source_parameters[ii+1]&&kk>=(int)source_parameters[ii+2]&&j<((int)source_parameters[ii]+(int)source_parameters[ii+3])&&i<((int)source_parameters[ii+1]+(int)source_parameters[ii+4])&&kk<((int)source_parameters[ii+2]+(int)source_parameters[ii+5]))
							{
								Told[i*(grid_height)*(grid_depth)+j*(grid_depth)+kk]=source_parameters[ii+6];
								break;
							}
							else
							{
								Told[i*(grid_height)*(grid_depth)+j*(grid_depth)+kk]=start_temp;
							}
						}
					}
				}
			}

		int p_n=paramter_number-7;
		float* sp = &source_parameters[0];

		hipMalloc((void **)&d_Told, size_T);
		hipMalloc((void **)&d_Tnew, size_T);
		hipMalloc((void **)&d_k_val,sizeof(float));
		hipMalloc((void **)&d_dim,sizeof(int));
		hipMalloc((void **)&d_sp,p_n*sizeof(float));
		hipMalloc((void **)&d_grid_w,sizeof(int));
		hipMalloc((void **)&d_grid_h,sizeof(int));
		hipMalloc((void **)&d_grid_d,sizeof(int));
		hipMalloc((void **)&pn,sizeof(int));

		hipMemcpy(d_Told,Told,size_T,hipMemcpyHostToDevice);
		hipMemcpy(d_Tnew,Tnew,size_T,hipMemcpyHostToDevice);
		hipMemcpy(d_k_val,&k,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d_dim,&dimension,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_sp,sp,p_n*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_w,&grid_width,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_h,&grid_height,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_grid_d,&grid_depth,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(pn,&p_n,sizeof(int),hipMemcpyHostToDevice);

		dim3 Block(BLOCK_SIZE,BLOCK_SIZE,BLOCK_SIZE);
		dim3 Grid((int)ceil((grid_width+BLOCK_SIZE-1)/BLOCK_SIZE),(int)ceil((grid_height+BLOCK_SIZE-1)/BLOCK_SIZE),(int)ceil((grid_depth+BLOCK_SIZE-1)/BLOCK_SIZE));

		for (int t = 0; t <timestep; t++)
		{	
			kernel<<<Grid,Block>>>(d_Told,d_Tnew,d_dim,d_k_val,d_sp,d_grid_w,d_grid_h,d_grid_d,pn);
		}
		hipMemcpy(Tnew,d_Tnew,size_T,hipMemcpyDeviceToHost);
		hipFree(d_Tnew);hipFree(d_Told);hipFree(d_k_val);hipFree(d_dim);hipFree(d_sp);hipFree(d_sp);
		hipFree(d_grid_d);hipFree(d_grid_h);hipFree(d_grid_w);

		/*cout<<"\nTnew: \n";
		for (int kk = 0; kk < grid_depth ; kk++)
		{
			cout<<"\n";
			for (int i = 0; i < grid_height; i++)
			{
				for (int j = 0; j < grid_width-1; j++)
				{
					cout<<left<<Tnew[i*(grid_height)*(grid_depth)+j*(grid_depth)+kk]<<", ";
					//printf("%f, ",Tnew[i][j] );
				}
				cout<<left<<Tnew[(i*grid_height + grid_width-1)*grid_depth+kk]<<endl;
				//printf("%f,\n",Tnew[i][grid_width-1] );
			}
			cout<<"\n";
		}*/
		ofstream build ("heatOutput.csv", std::ofstream::out);
		for (int kk = 0; kk < grid_depth; kk++)
		{
			for (int i = 0; i < grid_height; i++)
			{
				for (int j = 0; j < grid_width-1; j++)
				{
					build<<std::setprecision(6)<<Tnew[i*(grid_height)*(grid_depth)+j*(grid_depth)+kk]<<", ";
				}
				build<<std::setprecision(6)<<Tnew[i*(grid_height)*(grid_depth)+(grid_width-1)*(grid_depth)+kk];
				if (kk!=grid_depth-1||i!=grid_height-1)
				{	
					build<<"\n";
				}
			}
			if (kk!=grid_depth-1)
			{
				build<<"\n";
			}
			
		}
		build.close();
	}
	return 0;
}